
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime_api.h>

using namespace std::chrono;

template<unsigned int blockSize>
__device__ void warpReduce(volatile float *sdata, unsigned int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template<unsigned int blockSize>
__global__ void reduce_final(float *g_idata, float *g_odata, unsigned int n) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockSize * 2) + tid;
    unsigned int gridSize = blockSize * 2 * gridDim.x;
    sdata[tid] = 0;
    while (i < n) {
        sdata[tid] += g_idata[i] + g_idata[i + blockSize];
        i += gridSize;
    }
    __syncthreads();
    if (blockSize >= 512) {
        if (tid < 256) { sdata[tid] += sdata[tid + 256]; }
        __syncthreads();
    }
    if (blockSize >= 256) {
        if (tid < 128) { sdata[tid] += sdata[tid + 128]; }
        __syncthreads();
    }
    if (blockSize >= 128) {
        if (tid < 64) { sdata[tid] += sdata[tid + 64]; }
        __syncthreads();
    }
    if (tid < 32) warpReduce<blockSize>(sdata, tid);
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


int main(void) {
    int N = 100000000;
    float *g_indata_host, *g_indata_device, *g_outdata_host, *g_outdata_device;
    g_indata_host = (float *) malloc(N * sizeof(float));
    g_outdata_host = (float *) malloc(sizeof(float));

    hipMalloc(&g_indata_device, N * sizeof(float));
    hipMalloc(&g_outdata_device, sizeof(float));

    for (auto i = 0; i < N; i++) {
        g_indata_host[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);;
    }

    hipMemcpy(g_indata_device, g_indata_host, N * sizeof(float), hipMemcpyHostToDevice);

//    This is where the code is run
    auto dimGrid = 512;
    auto dimBlock = 512;
    auto smemSize = 128 * sizeof(float);
    auto threads = 512;
    auto start = high_resolution_clock::now();
    auto n = 2048;
    switch (threads) {
        case 512:
            reduce_final<512><<<dimGrid, dimBlock, smemSize>>>(g_indata_device, g_outdata_device, n);
            break;
        case 256:
            reduce_final<256><<<dimGrid, dimBlock, smemSize>>>(g_indata_device, g_outdata_device, n);
            break;
        case 128:
            reduce_final<128><<<dimGrid, dimBlock, smemSize>>>(g_indata_device, g_outdata_device, n);
            break;
        case 64:
            reduce_final<64><<<dimGrid, dimBlock, smemSize>>>(g_indata_device, g_outdata_device, n);
            break;
        case 32:
            reduce_final<32><<<dimGrid, dimBlock, smemSize>>>(g_indata_device, g_outdata_device, n);
            break;
        case 16:
            reduce_final<16><<<dimGrid, dimBlock, smemSize>>>(g_indata_device, g_outdata_device, n);
            break;
        case 8:
            reduce_final<8><<<dimGrid, dimBlock, smemSize>>>(g_indata_device, g_outdata_device, n);
            break;
        case 4:
            reduce_final<4><<<dimGrid, dimBlock, smemSize>>>(g_indata_device, g_outdata_device, n);
            break;
        case 2:
            reduce_final<2><<<dimGrid, dimBlock, smemSize>>>(g_indata_device, g_outdata_device, n);
            break;
        case 1:
            reduce_final<1><<<dimGrid, dimBlock, smemSize>>>(g_indata_device, g_outdata_device, n);
            break;
    }

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    std::cout << "Time taken by function: "
              << duration.count() << " microseconds" << std::endl;
    hipFree(g_indata_device);
    hipFree(g_outdata_device);
    free(g_indata_host);
    free(g_outdata_host);

}