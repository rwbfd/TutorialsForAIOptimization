#include "hip/hip_runtime.h"
template <unsigned int blockSize>
__device__ void warp_reduce(volatile int *sdata, unsigned int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize>
__global__ void reduce(int *g_idata, int *g_odata, unsigned int n) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + tid;
    unsigned int gridSize = blockSize*2*gridDim.x;
    sdata[tid] = 0;
    while (i < n) { sdata[tid] += g_idata[i] + g_idata[i+blockSize]; i += gridSize; }
    __syncthreads();
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
    if (tid < 32) warp_reduce(sdata, tid);
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__host__ __device__ void launch_kernel(int dimGrid, int dimBlock, unsigned  long smemSize, *int a, *int c){
    switch (threads)
    {
        case 512:
            reduce<<<dimGrid, dimBlock, smemSize>>>(a, c); break;
        case 256:
            reduce<256><<<dimGrid, dimBlock, smemSize>>>(a, c); break;
        case 128:
            reduce<128><<<dimGrid, dimBlock, smemSize>>>(a, c); break;
        case 64:
            reduce<64><<<dimGrid, dimBlock, smemSize>>>(a, c); break;
        case 32:
            reduce<32><<<dimGrid, dimBlock, smemSize>>>(a, c); break;
        case 16:
            reduce<16><<<dimGrid, dimBlock, smemSize>>>(a, c); break;
        case 8:
            reduce<8><<<dimGrid, dimBlock, smemSize>>>(a, c); break;
        case 4:
            reduce<4><<<dimGrid, dimBlock, smemSize>>>(a, c); break;
        case 2:
            reduce<2><<<dimGrid, dimBlock, smemSize>>>(a, c); break;
        case 1:
            reduce<1><<<dimGrid, dimBlock, smemSize>>>(a, c); break;
    }
}